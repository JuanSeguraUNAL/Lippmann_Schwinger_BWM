#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <chrono>
#include <limits>
#include "ConfocalParabolicBilliard.cuh"
#include "BoundaryWallMethod.cuh"

int main(int argc, char* argv[]){
    if(argc<6){
        std::cerr<<"Uso: "<<argv[0]<<" xi0 eta0 k2\n";
        return 1;
    }
    double xi0   = std::stod(argv[1]);
    double eta0  = std::stod(argv[2]);
    double k2    = std::stod(argv[3]);
    double angle = std::stod(argv[4]);
    double gamma = std::numeric_limits<double>::infinity();
    int    num_b = std::stod(argv[5]);
    int    Ngrid = num_b;

    std::chrono::high_resolution_clock::time_point start, end;
    std::chrono::duration<double> elapsed;

    // 1) Frontera
    bwm::ConfocalParabolicBilliard billiard(xi0, eta0, num_b);
    auto boundary = billiard.getBoundary();
    std::ofstream foutB("boundary.dat");
    for(int i = 0; i < num_b; i++){
        foutB << boundary[i].x << "\t" << boundary[i].y << "\n";
    }
    foutB.close();
    hipDeviceSynchronize();

    // 2) Solver BWM
    double* gammas = (double*)malloc(sizeof(double));
    gammas[0] = gamma;
    bwm::BoundaryWallMethod solver(boundary, num_b, gammas, 1, std::sqrt(k2), angle);
    hipDeviceSynchronize();

    // 3) Malla y volcado de density.dat y phase.dat
    start = std::chrono::high_resolution_clock::now();
    std::ofstream foutD("density.dat"), foutP("phase.dat");
    double xmin=-8, xmax=8;
    Point *P = (Point*)malloc(sizeof(Point));
    for(int i = 0; i < Ngrid; ++i){
        double y = xmin + i*(xmax - xmin) / (Ngrid - 1);
        for(int j = 0; j < Ngrid; ++j){
            double x = xmin + j*(xmax-xmin)/(Ngrid-1);
            Point p = {x,y};
            P[0] = p;
            //std::cout << "INICIO PUNTO (" << x << " , " << y << ")\n";
            auto psi = solver.computeScatteredWave(P, Ngrid)[0];
            hipDeviceSynchronize();
            foutD << (psi.x * psi.x + psi.y * psi.y) << (j+1==Ngrid? "\n":" ");
            foutP << atan2(psi.y, psi.x) << (j+1==Ngrid? "\n":" ");
            //std::cout << "FIN PUNTO (" << x << " , " << y << ")\n";
        }
    }

    end = std::chrono::high_resolution_clock::now();
    elapsed = end - start;

    std::cout << num_b << "\t" << elapsed.count() << "\n";

    return 0;
}