#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <limits>
#include "ConfocalParabolicBilliard.cuh"
#include "BoundaryWallMethod.cuh"

int main(int argc, char* argv[]){
    if(argc<4){
        std::cerr<<"Uso: "<<argv[0]<<" xi0 eta0 k2\n";
        return 1;
    }
    double xi0   = std::stod(argv[1]);
    double eta0  = std::stod(argv[2]);
    double k2    = std::stod(argv[3]);
    double angle_deg = std::stod(argv[4]);
    double angle = angle_deg * (M_PI / 180.0);
    double gamma = std::numeric_limits<double>::infinity();
    int    num_b = 200;
    int    Ngrid = 200;

    // 1) Frontera
    bwm::ConfocalParabolicBilliard billiard(xi0, eta0, num_b);
    auto boundary = billiard.getBoundary();
    std::ofstream foutB("boundary.dat");
    for(int i = 0; i < num_b; i++){
        foutB << boundary[i].x << "\t" << boundary[i].y << "\n";
    }
    foutB.close();
    hipDeviceSynchronize();

    // 2) Solver BWM
    double* gammas = (double*)malloc(sizeof(double));
    gammas[0] = gamma;
    bwm::BoundaryWallMethod solver(boundary, num_b, gammas, 1, std::sqrt(k2), angle);
    hipDeviceSynchronize();

    // 3) Malla y volcado de density.dat y phase.dat
    std::ofstream foutD("density.dat"), foutP("phase.dat");
    double xmin=-8, xmax=8;
    Point *P = (Point*)malloc(sizeof(Point));
    for(int i = 0; i < Ngrid; ++i){
        double y = xmin + i*(xmax - xmin) / (Ngrid - 1);
        for(int j = 0; j < Ngrid; ++j){
            double x = xmin + j*(xmax-xmin)/(Ngrid-1);
            Point p = {x,y};
            P[0] = p;
            auto psi = solver.computeScatteredWave(P, Ngrid)[0];
            hipDeviceSynchronize();
            foutD << (psi.x * psi.x + psi.y * psi.y) << (j+1==Ngrid? "\n":" ");
            foutP << atan2(psi.y, psi.x) << (j+1==Ngrid? "\n":" ");
        }
    }
    return 0;
}